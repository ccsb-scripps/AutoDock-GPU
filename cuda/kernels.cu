#include "hip/hip_runtime.h"
/*

AutoDock-GPU, an OpenCL implementation of AutoDock 4.2 running a Lamarckian Genetic Algorithm
Copyright (C) 2017 TU Darmstadt, Embedded Systems and Applications Group, Germany. All rights reserved.
For some of the code, Copyright (C) 2019 Computational Structural Biology Center, the Scripps Research Institute.

AutoDock is a Trade Mark of the Scripps Research Institute.

This library is free software; you can redistribute it and/or
modify it under the terms of the GNU Lesser General Public
License as published by the Free Software Foundation; either
version 2.1 of the License, or (at your option) any later version.

This library is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
Lesser General Public License for more details.

You should have received a copy of the GNU Lesser General Public
License along with this library; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA

*/


#include <cstdint>
#include <cassert>
#include "defines.h"
#include "calcenergy.h"
#include "GpuData.h"

__device__ inline uint64_t llitoulli(int64_t l)
{
	uint64_t u;
	asm("mov.b64    %0, %1;" : "=l"(u) : "l"(l));
	return u;
}

__device__ inline int64_t ullitolli(uint64_t u)
{
	int64_t l;
	asm("mov.b64    %0, %1;" : "=l"(l) : "l"(u));
	return l;
}


#define WARPMINIMUMEXCHANGE(tgx, v0, k0, mask) \
	{ \
		float v1    = v0; \
		int k1      = k0; \
		int otgx    = tgx ^ mask; \
		float v2    = __shfl_sync(0xffffffff, v0, otgx); \
		int k2      = __shfl_sync(0xffffffff, k0, otgx); \
		int flag    = ((v1 < v2) ^ (tgx > otgx)) && (v1 != v2); \
		k0          = flag ? k1 : k2; \
		v0          = flag ? v1 : v2; \
	}

#define WARPMINIMUM2(tgx, v0, k0) \
	WARPMINIMUMEXCHANGE(tgx, v0, k0, 1) \
	WARPMINIMUMEXCHANGE(tgx, v0, k0, 2) \
	WARPMINIMUMEXCHANGE(tgx, v0, k0, 4) \
	WARPMINIMUMEXCHANGE(tgx, v0, k0, 8) \
	WARPMINIMUMEXCHANGE(tgx, v0, k0, 16)

#define REDUCEINTEGERSUM(value, pAccumulator) \
	if (threadIdx.x == 0) \
	{ \
		*pAccumulator = 0; \
	} \
	__threadfence(); \
	__syncthreads(); \
	if (__any_sync(0xffffffff, value != 0)) \
	{ \
		uint32_t tgx            = threadIdx.x & cData.warpmask; \
		value                  += __shfl_sync(0xffffffff, value, tgx ^ 1); \
		value                  += __shfl_sync(0xffffffff, value, tgx ^ 2); \
		value                  += __shfl_sync(0xffffffff, value, tgx ^ 4); \
		value                  += __shfl_sync(0xffffffff, value, tgx ^ 8); \
		value                  += __shfl_sync(0xffffffff, value, tgx ^ 16); \
		if (tgx == 0) \
		{ \
			atomicAdd(pAccumulator, value); \
		} \
	} \
	__threadfence(); \
	__syncthreads(); \
	value = *pAccumulator; \
	__syncthreads();

#define ATOMICADDI32(pAccumulator, value) atomicAdd(pAccumulator, (value))
#define ATOMICSUBI32(pAccumulator, value) atomicAdd(pAccumulator, -(value))
#define ATOMICADDF32(pAccumulator, value) atomicAdd(pAccumulator, (value))
#define ATOMICSUBF32(pAccumulator, value) atomicAdd(pAccumulator, -(value))

/* Begin: Reduction using tensor units */
/*
 * Half-precision support
 * https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH____HALF__MISC.html
 */
#include <hip/hip_fp16.h>

/*
 * Tensor Cores
 * https://developer.nvidia.com/blog/programming-tensor-cores-cuda-9
 *
 * Don't forget to compile specifying the architecture, e.g., sm_86.
 * For AutoDock-GPU, this can be done via the TARGETS option.
 * make DEVICE=GPU TESTLS=ad NUMWI=64 TARGETS=86 test
 * https://stackoverflow.com/a/53634598/1616865
 */
#include <mma.h>
using namespace nvcuda;

#define TILE_SIZE (16 * 16)

constexpr int rowscols_M = 16;	// Number of rows (or cols) in the M dimension
constexpr int rowscols_N = 16;	// Number of rows (or cols) in the N dimension
constexpr int rowscols_K = 16;	// Number of rows (or cols) in the K dimension

// Half constants
// HIPRT_ONE_FP16 was not recognized by the NVCC compiler
// So its value is indicated explicitly
// https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__INTRINSIC__HALF__CONSTANTS.html#group__CUDA__MATH__INTRINSIC__HALF__CONSTANTS
#define HALF_ONE __ushort_as_half((unsigned short)0x3C00U)
#define HALF_ZERO __ushort_as_half((unsigned short)0x0000U)

__device__ void fill_Q(half *Q_data) {

	half I4[16] = {
		HALF_ONE, HALF_ZERO, HALF_ZERO, HALF_ZERO,
		HALF_ZERO, HALF_ONE, HALF_ZERO, HALF_ZERO,
		HALF_ZERO, HALF_ZERO, HALF_ONE, HALF_ZERO,
		HALF_ZERO, HALF_ZERO, HALF_ZERO, HALF_ONE
	};

	/*
	// Naive implementation: a single thread fills data in
	if (threadIdx.x == 0) {
		for (uint i = 0; i < 4; i++) {	// How many rows (of 4x4 blocks) are there in matrix A?
			for (uint j = 0; j < 4; j++) {	// How many cols (of 4x4 blocks) are there in matrix A?
				for (uint ii = 0; ii < 4; ii++) {
					for (uint jj = 0; jj < 4; jj++) {
						Q_data[4*i + 64*j + ii + 16*jj] = I4 [4*ii + jj];
					}
				}
			}
		}
	}
	*/

	// Slightly improved multi-threaded implementation
	for (uint i = threadIdx.x; i < 4; i+=blockDim.x) {	// How many rows (of 4x4 blocks) are there in matrix A?
		for (uint j = 0; j < 4; j++) {	// How many cols (of 4x4 blocks) are there in matrix A?
			for (uint ii = 0; ii < 4; ii++) {
				for (uint jj = 0; jj < 4; jj++) {
					Q_data[4*i + 64*j + ii + 16*jj] = I4 [4*ii + jj];
				}
			}
		}
	}

	/*
	// Further improved multi-threaded implementation
	// (It didn't provide significant performance improvements -> commented out)
	// Fusing two outer loops into a single one
	// To do that: coeffs = 4i + 64j
	constexpr uint coeffs [16] = {0, 64, 128, 192, 4, 68, 132, 196, 8, 72, 136, 200, 12, 76, 140, 204};
	for (uint k = threadIdx.x; k < 16; k+=blockDim.x) {
		for (uint ii = 0; ii < 4; ii++) {
			for (uint jj = 0; jj < 4; jj++) {
				Q_data[coeffs[k] + ii + 16*jj] = I4 [4*ii + jj];
			}
		}	
	}
	*/

	/*
	// Enable this block to print matrix values
	if (blockIdx.x == 0 && threadIdx.x == 0) {
		printf("\nQ_data");
		for (uint i = 0; i < 16 * 16; i++) {
			if ((i % 16) == 0) {printf("\n[Row %u]: ", i/16);}
			printf(" %2.2f ", __half2float(Q_data[i]));
		}
		printf("\n");
    }
	*/
}

// Implementation based on M.Sc. thesis by Gabin Schieffer at KTH:
// "Accelerating a Molecular Docking Application by Leveraging Modern Heterogeneous Computing Systemx"
// https://www.diva-portal.org/smash/get/diva2:1786161/FULLTEXT01.pdf
__device__ void reduce_via_tensor_units(half *data_to_be_reduced) {

	__syncthreads();

	if (threadIdx.x <= 31) { // Only one warp performs reduction
		__shared__ __align__ (256) half Q_data[TILE_SIZE];

		fill_Q(Q_data);

		__shared__ __align__ (256) half tmp[TILE_SIZE];

		// Declaring and filling fragments - Those are *not* shared
		wmma::fragment<wmma::matrix_b, rowscols_M, rowscols_N, rowscols_K, half, wmma::col_major> frag_P;
		wmma::fragment<wmma::accumulator, rowscols_M, rowscols_N, rowscols_K, half> frag_V;

		wmma::fragment<wmma::matrix_a, rowscols_M, rowscols_N, rowscols_K, half, wmma::col_major> frag_Q;
		wmma::fragment<wmma::matrix_b, rowscols_M, rowscols_N, rowscols_K, half, wmma::col_major> frag_W;
		wmma::fragment<wmma::accumulator, rowscols_M, rowscols_N, rowscols_K, half> frag_C;

		wmma::fill_fragment(frag_P, HALF_ONE); // P: only ones
		wmma::fill_fragment(frag_V, HALF_ZERO); // Output: initialize to zeros
		wmma::fill_fragment(frag_C, HALF_ZERO); // Final result
		wmma::load_matrix_sync(frag_Q, Q_data, 16);

		// 1. Accumulate the values: V <- AP + V
		for(uint i = 0; i < (4 * NUM_OF_THREADS_PER_BLOCK)/TILE_SIZE; i++){
			const unsigned int offset = i * TILE_SIZE;
			wmma::fragment<wmma::matrix_a, rowscols_M, rowscols_N, rowscols_K, half, wmma::col_major> frag_A;
			wmma::load_matrix_sync(frag_A, data_to_be_reduced + offset, 16);
			wmma::mma_sync(frag_V, frag_A, frag_P, frag_V);
		}

		// W <- V (required since we need V as a "wmma::matrix_b")
		wmma::store_matrix_sync(tmp, frag_V, 16, wmma::mem_col_major);
		wmma::load_matrix_sync(frag_W, tmp, 16);

		// 2. Perform line sum: C <- QW + C (zero)
		wmma::mma_sync(frag_C, frag_Q, frag_W, frag_C);

		// 3. Store result in shared memory
		wmma::store_matrix_sync(data_to_be_reduced, frag_C, 16, wmma::mem_col_major);
	}

	__syncthreads();
}

/* End: Reduction using tensor units */

#define REDUCEFLOATSUM(value, pAccumulator) \
	if (threadIdx.x == 0) \
	{ \
		*pAccumulator = 0; \
	} \
	__threadfence(); \
	__syncthreads(); \
	if (__any_sync(0xffffffff, value != 0.0f)) \
	{ \
		uint32_t tgx            = threadIdx.x & cData.warpmask; \
		value                  += __shfl_sync(0xffffffff, value, tgx ^ 1); \
		value                  += __shfl_sync(0xffffffff, value, tgx ^ 2); \
		value                  += __shfl_sync(0xffffffff, value, tgx ^ 4); \
		value                  += __shfl_sync(0xffffffff, value, tgx ^ 8); \
		value                  += __shfl_sync(0xffffffff, value, tgx ^ 16); \
		if (tgx == 0) \
		{ \
			atomicAdd(pAccumulator, value); \
		} \
	} \
	__threadfence(); \
	__syncthreads(); \
	value = (float)(*pAccumulator); \
	__syncthreads();



static __constant__ GpuData cData;
static GpuData cpuData;

void SetKernelsGpuData(GpuData* pData)
{
	hipError_t status;
	status = hipMemcpyToSymbol(HIP_SYMBOL(cData), pData, sizeof(GpuData));
	RTERROR(status, "SetKernelsGpuData copy to cData failed");
	memcpy(&cpuData, pData, sizeof(GpuData));
}

void GetKernelsGpuData(GpuData* pData)
{
	hipError_t status;
	status = hipMemcpyFromSymbol(pData, HIP_SYMBOL(cData), sizeof(GpuData));
	RTERROR(status, "GetKernelsGpuData copy From cData failed");
}


// Kernel files
#include "calcenergy.cu"
#include "calcMergeEneGra.cu"
#include "auxiliary_genetic.cu"
#include "kernel1.cu"
#include "kernel2.cu"
#include "kernel3.cu"
#include "kernel4.cu"
#include "kernel_ad.cu"
#include "kernel_adam.cu"
