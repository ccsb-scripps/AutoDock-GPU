#include "hip/hip_runtime.h"
/*

AutoDock-GPU, an OpenCL implementation of AutoDock 4.2 running a Lamarckian Genetic Algorithm
Copyright (C) 2017 TU Darmstadt, Embedded Systems and Applications Group, Germany. All rights reserved.
For some of the code, Copyright (C) 2019 Computational Structural Biology Center, the Scripps Research Institute.

AutoDock is a Trade Mark of the Scripps Research Institute.

This library is free software; you can redistribute it and/or
modify it under the terms of the GNU Lesser General Public
License as published by the Free Software Foundation; either
version 2.1 of the License, or (at your option) any later version.

This library is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
Lesser General Public License for more details.

You should have received a copy of the GNU Lesser General Public
License along with this library; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA

*/


//#define DEBUG_ENERGY_KERNEL

#define invpi2 1.0f/(PI_TIMES_2)

// Magic positive integer exponent power ... -AT
__forceinline__ __device__ float positive_power(float a, uint exp)
{
	float result=(exp & 1)?a:1.0f;
	while(exp>>=1){
		a *= a;
		result=(exp & 1)?result*a:result;
	}
	return result;
}

__forceinline__ __device__ float fmod_pi2(float x)
{
	return x-(int)(invpi2*x)*PI_TIMES_2;
}

#define fast_acos_a  9.78056e-05f
#define fast_acos_b -0.00104588f
#define fast_acos_c  0.00418716f
#define fast_acos_d -0.00314347f
#define fast_acos_e  2.74084f
#define fast_acos_f  0.370388f
#define fast_acos_o -(fast_acos_a+fast_acos_b+fast_acos_c+fast_acos_d)

__forceinline__ __device__ float fast_acos(float cosine)
{
	float x=fabs(cosine);
	float x2=x*x;
	float x3=x2*x;
	float x4=x3*x;
	float ac=(((fast_acos_o*x4+fast_acos_a)*x3+fast_acos_b)*x2+fast_acos_c)*x
	         +fast_acos_d
	         +fast_acos_e*sqrt(2.0f-sqrt(2.0f+2.0f*x))
	         -fast_acos_f*sqrt(2.0f-2.0f*x);
	return copysign(ac,cosine) + (cosine<0.0f)*PI_FLOAT;
}

__forceinline__ __device__ float4 cross(float3& u, float3& v)
{
	float4 result;
	result.x = u.y * v.z - v.y * u.z;
	result.y = v.x * u.z - u.x * v.z;
	result.z = u.x * v.y - v.x * u.y;
	result.w = 0.0f;
	return result;
}

__forceinline__ __device__ float4 cross(float4& u, float4& v)
{
	float4 result;
	result.x = u.y * v.z - v.y * u.z;
	result.y = v.x * u.z - u.x * v.z;
	result.z = u.x * v.y - v.x * u.y;
	result.w = 0.0f;
	return result;
}

__forceinline__ __device__ float4 quaternion_multiply(float4 a, float4 b)
{
	float4 result = { a.w * b.x + a.x * b.w + a.y * b.z - a.z * b.y, // x
			  a.w * b.y - a.x * b.z + a.y * b.w + a.z * b.x, // y
			  a.w * b.z + a.x * b.y - a.y * b.x + a.z * b.w, // z
			  a.w * b.w - a.x * b.x - a.y * b.y - a.z * b.z }; // w
	return result;
}
__forceinline__ __device__ float4 quaternion_rotate(float4 v, float4 rot)
{
	float4 result;
	
	float4 z = cross(rot,v);
	z.x *= 2.0f;
	z.y *= 2.0f;
	z.z *= 2.0f;
	float4 c = cross(rot, z);
	result.x = v.x + z.x * rot.w + c.x;
	result.y = v.y + z.y * rot.w + c.y;
	result.z = v.z + z.z * rot.w + c.z;
	result.w = 0.0f;
	return result;
}


// All related pragmas are in defines.h (accesible by host and device code)

__device__ void gpu_calc_energy(
                                float*  pGenotype,
                                float&  energy,
                                int&    run_id,
                                float3* calc_coords,
                                float*  pFloatAccumulator
                               )
// The GPU device function calculates the energy of the entity described by genotype, dockpars and the liganddata
// arrays in constant memory and returns it in the energy parameter. The parameter run_id has to be equal to the ID
// of the run whose population includes the current entity (which can be determined with blockIdx.x), since this
// determines which reference orientation should be used.
{
	energy = 0.0f;
#if defined (DEBUG_ENERGY_KERNEL)
	float interE = 0.0f;
	float intraE = 0.0f;
#endif

	// Initializing gradients (forces)
	// Derived from autodockdev/maps.py
	for (uint atom_id = threadIdx.x;
		  atom_id < cData.dockpars.num_of_atoms;
		  atom_id+= blockDim.x) {
		// Initialize coordinates
		calc_coords[atom_id].x = cData.pKerconst_conform->ref_coords_const[3*atom_id];
		calc_coords[atom_id].y = cData.pKerconst_conform->ref_coords_const[3*atom_id+1];
		calc_coords[atom_id].z = cData.pKerconst_conform->ref_coords_const[3*atom_id+2];
	}

	// General rotation moving vector
	float4 genrot_movingvec;
	genrot_movingvec.x = pGenotype[0];
	genrot_movingvec.y = pGenotype[1];
	genrot_movingvec.z = pGenotype[2];
	genrot_movingvec.w = 0.0f;
	// Convert orientation genes from sex. to radians
	float phi         = pGenotype[3] * DEG_TO_RAD;
	float theta       = pGenotype[4] * DEG_TO_RAD;
	float genrotangle = pGenotype[5] * DEG_TO_RAD;

	float4 genrot_unitvec;
	float sin_angle = sin(theta);
	float s2 = sin(genrotangle * 0.5f);
	genrot_unitvec.x = s2*sin_angle*cos(phi);
	genrot_unitvec.y = s2*sin_angle*sin(phi);
	genrot_unitvec.z = s2*cos(theta);
	genrot_unitvec.w = cos(genrotangle*0.5f);

	uint g1 = cData.dockpars.gridsize_x;
	uint g2 = cData.dockpars.gridsize_x_times_y;
	uint g3 = cData.dockpars.gridsize_x_times_y_times_z;

	__syncthreads();

	// ================================================
	// CALCULATING ATOMIC POSITIONS AFTER ROTATIONS
	// ================================================
	for (uint rotation_counter  = threadIdx.x;
	          rotation_counter  < cData.dockpars.rotbondlist_length;
	          rotation_counter += blockDim.x)
	{
		int rotation_list_element = cData.pKerconst_rotlist->rotlist_const[rotation_counter];

		if ((rotation_list_element & RLIST_DUMMY_MASK) == 0) // If not dummy rotation
		{
			uint atom_id = rotation_list_element & RLIST_ATOMID_MASK;

			// Capturing atom coordinates
			float4 atom_to_rotate;
			atom_to_rotate.x = calc_coords[atom_id].x;
			atom_to_rotate.y = calc_coords[atom_id].y;
			atom_to_rotate.z = calc_coords[atom_id].z;
			atom_to_rotate.w = 0.0f;

			// initialize with general rotation values
			float4 rotation_unitvec;
			float4 rotation_movingvec;
			if (atom_id < cData.dockpars.true_ligand_atoms){
				rotation_unitvec = genrot_unitvec;
				rotation_movingvec = genrot_movingvec;
			} else{
				rotation_unitvec.x = 0.0f; rotation_unitvec.y = 0.0f; rotation_unitvec.z = 0.0f;
				rotation_unitvec.w = 1.0f;
				rotation_movingvec.x = 0.0f; rotation_movingvec.y = 0.0f; rotation_movingvec.z = 0.0f;
				rotation_movingvec.w = 0.0f;
			}

			if ((rotation_list_element & RLIST_GENROT_MASK) == 0) // If rotating around rotatable bond
			{
				uint rotbond_id = (rotation_list_element & RLIST_RBONDID_MASK) >> RLIST_RBONDID_SHIFT;

				float rotation_angle = pGenotype[6+rotbond_id]*DEG_TO_RAD*0.5f;
				float s = sin(rotation_angle);
				rotation_unitvec.x = s*cData.pKerconst_conform->rotbonds_unit_vectors_const[3*rotbond_id];
				rotation_unitvec.y = s*cData.pKerconst_conform->rotbonds_unit_vectors_const[3*rotbond_id+1];
				rotation_unitvec.z = s*cData.pKerconst_conform->rotbonds_unit_vectors_const[3*rotbond_id+2];
				rotation_unitvec.w = cos(rotation_angle);
				rotation_movingvec.x = cData.pKerconst_conform->rotbonds_moving_vectors_const[3*rotbond_id];
				rotation_movingvec.y = cData.pKerconst_conform->rotbonds_moving_vectors_const[3*rotbond_id+1];
				rotation_movingvec.z = cData.pKerconst_conform->rotbonds_moving_vectors_const[3*rotbond_id+2];
				// Performing additionally the first movement which
				// is needed only if rotating around rotatable bond
				atom_to_rotate.x -= rotation_movingvec.x;
				atom_to_rotate.y -= rotation_movingvec.y;
				atom_to_rotate.z -= rotation_movingvec.z;
			}

			// Performing rotation and final movement
			float4 qt = quaternion_rotate(atom_to_rotate, rotation_unitvec);
			calc_coords[atom_id].x = qt.x + rotation_movingvec.x;
			calc_coords[atom_id].y = qt.y + rotation_movingvec.y;
			calc_coords[atom_id].z = qt.z + rotation_movingvec.z;
		} // End if-statement not dummy rotation

			__syncthreads();

	} // End rotation_counter for-loop

	// ================================================
	// CALCULATING INTERMOLECULAR ENERGY
	// ================================================
	float weights[8];
	float cube[8];
	for (uint atom_id = threadIdx.x;
	          atom_id < cData.dockpars.num_of_atoms;
	          atom_id+= blockDim.x)
	{
		if (cData.pKerconst_interintra->ignore_inter_const[atom_id]>0) // first two atoms of a flex res are to be ignored here
			continue;
		float x = calc_coords[atom_id].x;
		float y = calc_coords[atom_id].y;
		float z = calc_coords[atom_id].z;
		float q = cData.pKerconst_interintra->atom_charges_const[atom_id];
		uint atom_typeid = cData.pKerconst_interintra->atom_types_map_const[atom_id];
		if ((x < 0) || (y < 0) || (z < 0) || (x >= cData.dockpars.gridsize_x-1)
		                                  || (y >= cData.dockpars.gridsize_y-1)
		                                  || (z >= cData.dockpars.gridsize_z-1)){
			energy += 16777216.0f; //100000.0f;
			continue; // get on with loop as our work here is done (we crashed into the walls)
		}
		// Getting coordinates
		float x_low  = floor(x);
		float y_low  = floor(y);
		float z_low  = floor(z);

		// Grid value at 000
		float* grid_value_000 = cData.pMem_fgrids + ((ulong)(x_low  + y_low*g1  + z_low*g2)<<2);

		float dx = x - x_low;
		float omdx = 1.0f - dx;
		float dy = y - y_low; 
		float omdy = 1.0f - dy;
		float dz = z - z_low;
		float omdz = 1.0f - dz;

		// Calculating interpolation weights
		weights [idx_000] = omdx*omdy*omdz;
		weights [idx_010] = omdx*dy*omdz;
		weights [idx_001] = omdx*omdy*dz;
		weights [idx_011] = omdx*dy*dz;
		weights [idx_100] = dx*omdy*omdz;
		weights [idx_110] = dx*dy*omdz;
		weights [idx_101] = dx*omdy*dz;
		weights [idx_111] = dx*dy*dz;

		ulong mul_tmp = atom_typeid*g3<<2;
		cube[0] = *(grid_value_000+mul_tmp+0);
		cube[1] = *(grid_value_000+mul_tmp+1);
		cube[2] = *(grid_value_000+mul_tmp+2);
		cube[3] = *(grid_value_000+mul_tmp+3);
		cube[4] = *(grid_value_000+mul_tmp+4);
		cube[5] = *(grid_value_000+mul_tmp+5);
		cube[6] = *(grid_value_000+mul_tmp+6);
		cube[7] = *(grid_value_000+mul_tmp+7);
		// Calculating affinity energy
		energy += cube[0]*weights[0] + cube[1]*weights[1] + cube[2]*weights[2] + cube[3]*weights[3] + cube[4]*weights[4] + cube[5]*weights[5] + cube[6]*weights[6] + cube[7]*weights[7];
		#if defined (DEBUG_ENERGY_KERNEL)
		interE += cube[0]*weights[0] + cube[1]*weights[1] + cube[2]*weights[2] + cube[3]*weights[3] + cube[4]*weights[4] + cube[5]*weights[5] + cube[6]*weights[6] + cube[7]*weights[7];
		#endif

		// Capturing electrostatic values
		atom_typeid = cData.dockpars.num_of_map_atypes;

		mul_tmp = atom_typeid*g3<<2; // different atom type id to get charge IA
		cube[0] = *(grid_value_000+mul_tmp+0);
		cube[1] = *(grid_value_000+mul_tmp+1);
		cube[2] = *(grid_value_000+mul_tmp+2);
		cube[3] = *(grid_value_000+mul_tmp+3);
		cube[4] = *(grid_value_000+mul_tmp+4);
		cube[5] = *(grid_value_000+mul_tmp+5);
		cube[6] = *(grid_value_000+mul_tmp+6);
		cube[7] = *(grid_value_000+mul_tmp+7);

		// Calculating affinity energy
		energy += q * (cube[0]*weights[0] + cube[1]*weights[1] + cube[2]*weights[2] + cube[3]*weights[3] + cube[4]*weights[4] + cube[5]*weights[5] + cube[6]*weights[6] + cube[7]*weights[7]);
		#if defined (DEBUG_ENERGY_KERNEL)
		interE += q *(cube[0]*weights[0] + cube[1]*weights[1] + cube[2]*weights[2] + cube[3]*weights[3] + cube[4]*weights[4] + cube[5]*weights[5] + cube[6]*weights[6] + cube[7]*weights[7]);
		#endif

		// Need only magnitude of charge from here on down
		q = fabs(q);
		// Capturing desolvation values (atom_typeid+1 compared to above => mul_tmp + g3*4)
		mul_tmp += g3<<2;
		cube[0] = *(grid_value_000+mul_tmp+0);
		cube[1] = *(grid_value_000+mul_tmp+1);
		cube[2] = *(grid_value_000+mul_tmp+2);
		cube[3] = *(grid_value_000+mul_tmp+3);
		cube[4] = *(grid_value_000+mul_tmp+4);
		cube[5] = *(grid_value_000+mul_tmp+5);
		cube[6] = *(grid_value_000+mul_tmp+6);
		cube[7] = *(grid_value_000+mul_tmp+7);

		// Calculating affinity energy
		energy += q * (cube[0]*weights[0] + cube[1]*weights[1] + cube[2]*weights[2] + cube[3]*weights[3] + cube[4]*weights[4] + cube[5]*weights[5] + cube[6]*weights[6] + cube[7]*weights[7]);
		#if defined (DEBUG_ENERGY_KERNEL)
		interE += q *(cube[0]*weights[0] + cube[1]*weights[1] + cube[2]*weights[2] + cube[3]*weights[3] + cube[4]*weights[4] + cube[5]*weights[5] + cube[6]*weights[6] + cube[7]*weights[7]);
		#endif
	} // End atom_id for-loop (INTERMOLECULAR ENERGY)

#if defined (DEBUG_ENERGY_KERNEL)
	REDUCEFLOATSUM(interE, pFloatAccumulator)
#endif

	// In paper: intermolecular and internal energy calculation
	// are independent from each other, -> NO BARRIER NEEDED
	// but require different operations,
	// thus, they can be executed only sequentially on the GPU.
	float delta_distance = 0.5f * cData.dockpars.smooth;
	float smoothed_distance;

	// ================================================
	// CALCULATING INTRAMOLECULAR ENERGY
	// ================================================
	for (uint contributor_counter = threadIdx.x;
	          contributor_counter < cData.dockpars.num_of_intraE_contributors;
	          contributor_counter += blockDim.x)
	{
		// Getting atom IDs
		uint32_t atom1_id = cData.pKerconst_intracontrib->intraE_contributors_const[2*contributor_counter];
		uint32_t atom2_id = cData.pKerconst_intracontrib->intraE_contributors_const[2*contributor_counter+1];

		// Calculating vector components of vector going
		// from first atom's to second atom's coordinates
		float subx = calc_coords[atom1_id].x - calc_coords[atom2_id].x;
		float suby = calc_coords[atom1_id].y - calc_coords[atom2_id].y;
		float subz = calc_coords[atom1_id].z - calc_coords[atom2_id].z;

		// Calculating atomic_distance
		float dist = sqrt(subx*subx + suby*suby + subz*subz);
		float atomic_distance = dist * cData.dockpars.grid_spacing;

		// Getting type IDs
		uint32_t atom1_typeid = cData.pKerconst_interintra->atom_types_const[atom1_id];
		uint32_t atom2_typeid = cData.pKerconst_interintra->atom_types_const[atom2_id];

		uint32_t atom1_type_vdw_hb = cData.pKerconst_intra->atom_types_reqm_const [atom1_typeid];
		uint32_t atom2_type_vdw_hb = cData.pKerconst_intra->atom_types_reqm_const [atom2_typeid];

		// ------------------------------------------------
		// Required only for flexrings
		// Checking if this is a CG-G0 atomic pair.
		// If so, then adding energy term (E = G_AD * distance).
		// Initial specification required NON-SMOOTHED distance.
		// This interaction is evaluated at any distance,
		// so no cuttoffs considered here!
		// vbond is G_AD when calculating flexrings, 0.0 otherwise
		float vbond = G_AD * (float)(((atom1_type_vdw_hb == ATYPE_CG_IDX) && (atom2_type_vdw_hb == ATYPE_G0_IDX)) ||
					  ((atom1_type_vdw_hb == ATYPE_G0_IDX) && (atom2_type_vdw_hb == ATYPE_CG_IDX)));
		energy += vbond * atomic_distance;
		// ------------------------------------------------

		// Calculating energy contributions
		// Cuttoff1: internuclear-distance at 8A only for vdw and hbond.
		if (atomic_distance < 8.0f)
		{
			uint32_t idx = atom1_typeid * cData.dockpars.num_of_atypes + atom2_typeid;
			ushort exps = cData.pKerconst_intra->VWpars_exp_const[idx];
			char m=(exps & 0xFF00)>>8;
			char n=(exps & 0xFF);
			// Getting optimum pair distance (opt_distance)
			float opt_distance = cData.pKerconst_intra->reqm_AB_const[idx];

			// Getting smoothed distance
			// smoothed_distance = function(atomic_distance, opt_distance)
			float opt_dist_delta = opt_distance - atomic_distance;
			if(fabs(opt_dist_delta)>=delta_distance){
				smoothed_distance = atomic_distance + copysign(delta_distance,opt_dist_delta);
			} else smoothed_distance = opt_distance;
			// Calculating van der Waals / hydrogen bond term
			energy += (cData.pKerconst_intra->VWpars_AC_const[idx]
			           -__powf(smoothed_distance,m-n)*cData.pKerconst_intra->VWpars_BD_const[idx])
			           *__powf(smoothed_distance,-m);
			#if defined (DEBUG_ENERGY_KERNEL)
			intraE += (cData.pKerconst_intra->VWpars_AC_const[idx]
			           -__powf(smoothed_distance,m-n)*cData.pKerconst_intra->VWpars_BD_const[idx])
			           *__powf(smoothed_distance,-m);
			#endif
		} // if cuttoff1 - internuclear-distance at 8A

		// Calculating energy contributions
		// Cuttoff2: internuclear-distance at 20.48A only for el and sol.
		if (atomic_distance < 20.48f)
		{
			if(atomic_distance<cData.dockpars.elec_min_distance)
				atomic_distance=cData.dockpars.elec_min_distance;
			float q1 = cData.pKerconst_interintra->atom_charges_const[atom1_id];
			float q2 = cData.pKerconst_interintra->atom_charges_const[atom2_id];
//			float exp_el = native_exp(DIEL_B_TIMES_H*atomic_distance);
			float dist2 = atomic_distance*atomic_distance;
			// Calculating desolvation term
			// 1/25.92 = 0.038580246913580245
			float desolv_energy =  ((cData.pKerconst_intra->dspars_S_const[atom1_typeid] +
						 cData.dockpars.qasp*fabs(q1)) * cData.pKerconst_intra->dspars_V_const[atom2_typeid] +
						(cData.pKerconst_intra->dspars_S_const[atom2_typeid] +
						 cData.dockpars.qasp*fabs(q2)) * cData.pKerconst_intra->dspars_V_const[atom1_typeid]) *
						 (
							cData.dockpars.coeff_desolv*(12.96f-0.1063f*dist2*(1.0f-0.001947f*dist2)) /
							(12.96f+dist2*(0.4137f+dist2*(0.00357f+0.000112f*dist2)))
						 );
			// Calculating electrostatic term
			float dist_shift=atomic_distance+1.26366f;
			dist2=dist_shift*dist_shift;
			float diel = (1.10859f / dist2)+0.010358f;
			float es_energy = cData.dockpars.coeff_elec * q1 * q2 / atomic_distance;
			energy += diel * es_energy + desolv_energy;

			#if defined (DEBUG_ENERGY_KERNEL)
			intraE += diel * es_energy + desolv_energy;
			#endif
		} // if cuttoff2 - internuclear-distance at 20.48A
	} // End contributor_counter for-loop (INTRAMOLECULAR ENERGY)

	// reduction to calculate energy
	REDUCEFLOATSUM(energy, pFloatAccumulator)
#if defined (DEBUG_ENERGY_KERNEL)
	REDUCEFLOATSUM(intraE, pFloatAccumulator)
#endif
}

